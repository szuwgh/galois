
#include <hip/hip_runtime.h>
extern "C" __global__ void matmul(float *A, float *B, float *C, int N)
{
    int ROW = blockIdx.y * blockDim.y + threadIdx.y;
    int COL = blockIdx.x * blockDim.x + threadIdx.x;

    float tmpSum = 0;

    if (ROW < N && COL < N)
    {
        // each thread computes one element of the block sub-matrix
        for (int i = 0; i < N; i++)
        {
            tmpSum += A[ROW * N + i] * B[i * N + COL];
        }
    }
    // printf(\"pos, (%d, %d) - N %d - value %d\\n\", ROW, COL, N, tmpSum);
    C[ROW * N + COL] = tmpSum;
}